#include<stdio.h>
#include<bits/stdc++.h>
#include<hip/hip_runtime.h>
#include<time.h>

using namespace std;

#define width 32

__global__ void optMult(int* a, int* b, int* ans,int m,int n, int p)
{
    __shared__ int tileA[width][width], tileB[width][width];

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int tx = threadIdx.x, ty = threadIdx.y;
    int temp_sum = 0;
    tileA[ty][tx] = 0, tileB[ty][tx] = 0;

    for(int i=0;i<(p+width-1)/width;i++)
    {
        if(row < m &&  (tx + i*width) < p)
            tileA[ty][tx] = a[row*p + i*width + tx];
        else
            tileA[ty][tx] = 0;

        if(col < n && (ty + i*width) < p)
            tileB[ty][tx] = b[col + (ty + i*width)*n ];
        else
            tileB[ty][tx] = 0;

        __syncthreads();

        for(int i=0;i<width;i++)
            temp_sum += tileA[ty][i]*tileB[i][tx];
    }

    if((row<m)&&(col<n))
    {
        ans[row*n + col] = temp_sum;
    }
}

void init(int* a,int* b,int m,int n,int p)
{
    for(int i=0;i<m;i++)
    {   
        for(int j=0;j<p;j++)
            *(a+i*p+j) = 1;
    }

    for(int i=0;i<p;i++)
    {   
        for(int j=0;j<n;j++)
             *(b+i*n+j) = 2;
    }
   
}

int main()
{
    int *h_a, *h_b, *h_c; 
    int *d_a, *d_b, *d_c; 
    
    int m,p,n;

    m = 4, p = 2, n = 4; 

    size_t bytes_a = m*p*sizeof(int);
    size_t bytes_b = n*p*sizeof(int);
    size_t bytes_c = m*n*sizeof(int);

    h_a = (int*)malloc(bytes_a);
    h_b = (int*)malloc(bytes_b);
    h_c = (int*)malloc(bytes_c);

    init(h_a,h_b,m,n,p);

    hipMalloc(&d_a,bytes_a);
    hipMalloc(&d_b,bytes_b);
    hipMalloc(&d_c,bytes_c);

    int block_size = 32;
    int grid_size = (int)ceil((float)64/block_size);

    dim3 grid(grid_size,grid_size);
    dim3 threads(block_size,block_size);

    hipMemcpy(d_a,h_a,bytes_a,hipMemcpyHostToDevice);
    hipMemcpy(d_b,h_b,bytes_b,hipMemcpyHostToDevice);
    hipMemcpy(d_c,h_c,bytes_c,hipMemcpyHostToDevice);

    optMult<<<grid,threads>>> (d_a,d_b,d_c,m,n,p);

    hipMemcpy(h_c,d_c,bytes_c,hipMemcpyDeviceToHost);

    for(int i=0;i<m;i++)
    {
        for(int j=0;j<n;j++)
            cout<<*(h_c+i*n+j)<<" ";
        cout<<"\n";
    }
    
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}
