#include<stdio.h>
#include<bits/stdc++.h>
#include<hip/hip_runtime.h>
#include<time.h>

using namespace std;

__global__ void matMul(int* A, int* B, int* C,int m,int n, int p)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int temp_sum = 0;
    if((row<m)&&(col<n))
    {
        for(int k=0;k<p;k++)
        {   
            temp_sum += A[row*p + k]*B[k*n + col];
        }
        C[row*n + col] = temp_sum;
    }
}

void init(int* A,int* B,int m,int n,int p)
{
    for(int i=0;i<m;i++)
    {   
        for(int j=0;j<p;j++)
            *(A+i*p+j) = 1;
    }

    for(int i=0;i<p;i++)
    {   
        for(int j=0;j<n;j++)
             *(B+i*n+j) = 2;
    }
   
}

int main()
{
    int *h_a, *h_b, *h_c; //host pointers
    int *d_a, *d_b, *d_c; //device pointers
    
    int m,p,n;

    m = 1000, p = 1000, n = 1000; 

    size_t bytes_a = m*p*sizeof(int);
    size_t bytes_b = n*p*sizeof(int);
    size_t bytes_c = m*n*sizeof(int);

    h_a = (int*)malloc(bytes_a);
    h_b = (int*)malloc(bytes_b);
    h_c = (int*)malloc(bytes_c);

    init(h_a,h_b,m,n,p);

    hipMalloc(&d_a,bytes_a);
    hipMalloc(&d_b,bytes_b);
    hipMalloc(&d_c,bytes_c);

    int block_size = 16;
    int grid_size = (int)ceil((float)32/block_size);

    

    dim3 grid(grid_size,grid_size);
    dim3 threads(block_size,block_size);

    // cout<<block_size<<" "<<grid_size<<"\n";

    hipMemcpy(d_a,h_a,bytes_a,hipMemcpyHostToDevice);
    hipMemcpy(d_b,h_b,bytes_b,hipMemcpyHostToDevice);
    hipMemcpy(d_c,h_c,bytes_c,hipMemcpyHostToDevice);

    

    matMul<<<grid,threads>>> (d_a,d_b,d_c,m,n,p);

    hipMemcpy(h_c,d_c,bytes_c,hipMemcpyDeviceToHost);

    //for(int i=0;i<m;i++)
    //{
    //    for(int j=0;j<n;j++)
    //        cout<<*(h_c+i*n+j)<<" ";
    //    cout<<"\n";
    //}
    
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}
